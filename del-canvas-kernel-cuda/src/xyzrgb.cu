#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "mat4_col_major.h"
#include "aabb2.h"

extern "C" {

struct XyzRgb{
    float xyz[3];
    unsigned char rgb[3];
};

struct Splat {
    float z;
    float pos_pix[2];
    float rad;
};

__global__
void xyzrgb_to_splat(
  uint32_t num_pnt,
  Splat* pnt2splat,
  const XyzRgb *pnt2xyzrgb,
  const float *transform_world2ndc,
  const uint32_t img_w,
  const uint32_t img_h,
  float radius)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const auto p0 = pnt2xyzrgb[i_pnt].xyz;
    const auto q0 = mat4_col_major::transform_homogeneous(
        transform_world2ndc, p0);
   float r0[2] = {
     (q0[0] + 1.f) * 0.5f * float(img_w),
     (1.f - q0[1]) * 0.5f * float(img_h) };
   float rad;
   {
       const cuda::std::array<float,9> dqdp = mat4_col_major::jacobian_transform(transform_world2ndc, p0);
       const cuda::std::array<float,9> dpdq = mat3_col_major::try_inverse(dqdp.data()).value();
       const float dx[3] = { dpdq[0], dpdq[1], dpdq[2] };
       const float dy[3] = { dpdq[3], dpdq[4], dpdq[5] };
       float rad_pix_x = (1.f / vec3::norm(dx)) * 0.5f * float(img_w) * radius;
       float rad_pxi_y = (1.f / vec3::norm(dy)) * 0.5f * float(img_h) * radius;
       rad = 0.5f * (rad_pix_x + rad_pxi_y);
   }
   pnt2splat[i_pnt].z = q0[2];
   pnt2splat[i_pnt].pos_pix[0] = r0[0];
   pnt2splat[i_pnt].pos_pix[1] = r0[1];
   pnt2splat[i_pnt].rad = rad;
}


__global__
void count_splat_in_tile(
  uint32_t num_pnt,
  const Splat* pnt2splat,
  uint32_t* tile2ind,
  uint32_t* pnt2ind,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat& splat = pnt2splat[i_pnt];
    const cuda::std::array<float,4> aabb = aabb2::from_point(splat.pos_pix, splat.rad);
    //
    float tile_size_f = float(tile_size);
    int ix0 = int(floor(aabb[0] / tile_size_f));
    int iy0 = int(floor(aabb[1] / tile_size_f));
    int ix1 = int(floor(aabb[2] / tile_size_f))+1;
    int iy1 = int(floor(aabb[3] / tile_size_f))+1;
    uint32_t cnt = 0;
    // printf("%d %d %d %d\n", ix0, iy0, ix1, iy1);
    for(int ix = ix0; ix < ix1; ++ix ) {
        if( ix < 0 || ix >= tile_w ){
            continue;
        }
        for(int iy=iy0;iy<iy1;++iy) {
            if( iy < 0 || iy >= tile_h ){
                continue;
            }
            int i_tile = iy * tile_w + ix;
            // printf("%d %d\n", i_pnt, i_tile);
            atomicAdd(&tile2ind[i_tile], 1);
            ++cnt;
        }
    }
    pnt2ind[i_pnt] = cnt;
}

__device__ uint32_t float_to_uint32(float value) {
    uint32_t result;
    memcpy(&result, &value, sizeof(result));
    return result;
}

__device__ uint64_t concatenate32To64(uint32_t a, uint32_t b) {
    // b を64ビットの下位部分に、a を64ビットの上位部分にシフトして結合
    return ((uint64_t)b) | (((uint64_t)a) << 32);
}

__global__
void fill_index_info(
  uint32_t num_pnt,
  const Splat* pnt2splat,
  const uint32_t* pnt2idx,
  uint64_t* idx2tiledepth,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat& splat = pnt2splat[i_pnt];
    const cuda::std::array<float,4> aabb = aabb2::from_point(splat.pos_pix, splat.rad);
    //
    float tile_size_f = float(tile_size);
    int ix0 = int(floor(aabb[0] / tile_size_f));
    int iy0 = int(floor(aabb[1] / tile_size_f));
    int ix1 = int(floor(aabb[2] / tile_size_f))+1;
    int iy1 = int(floor(aabb[3] / tile_size_f))+1;
    uint32_t cnt = 0;
    // printf("%d %d %d %d\n", ix0, iy0, ix1, iy1);
    for(int ix = ix0; ix < ix1; ++ix ) {
        if( ix < 0 || ix >= tile_w ){
            continue;
        }
        for(int iy=iy0;iy<iy1;++iy) {
            if( iy < 0 || iy >= tile_h ){
                continue;
            }
            uint32_t i_tile = iy * tile_w + ix;
            uint32_t depth_in_uint32 = float_to_uint32(splat.z);
            uint64_t tiledepth= concatenate32To64(i_tile, depth_in_uint32);
            idx2tiledepth[pnt2idx[i_pnt] + cnt] = tiledepth;
            ++cnt;
        }
    }
    // pnt2ind[i_pnt] = cnt;
}



} // extern "C"