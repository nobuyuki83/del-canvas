#include "hip/hip_runtime.h"
#include "mat2_sym.h"
#include "mat4_col_major.h"
#include "mat2x3_col_major.h"
#include "quaternion.h"
#include "aabb2.h"

extern "C" {

struct Splat3 {
    float xyz[3];
    float rgb_dc[3];
    float rgb_sh[45];
    float opacity;
    float scale[3];
    float quaternion[4];
};

struct Splat2 {
    float pos_pix[2];
    float sig_inv[3];
    float aabb[4];
    float rgb[3];
    float ndc_z;
};

__global__
void splat3_to_splat2(
  uint32_t num_pnt,
  Splat2* pnt2splat2,
  const Splat3 *pnt2splat3,
  const float *transform_world2ndc,
  const uint32_t img_w,
  const uint32_t img_h)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const auto pos_world = pnt2splat3[i_pnt].xyz;
    const cuda::std::array<float,9> world2ndc = mat4_col_major::jacobian_transform(transform_world2ndc, pos_world);
    const cuda::std::array<float,6> ndc2pix = mat2x3_col_major::transform_ndc2pix(img_w, img_h);
    const cuda::std::array<float,6> world2pix = mat2x3_col_major::mult_mat3_col_major(ndc2pix.data(), world2ndc.data());
    const auto pos_ndc = mat4_col_major::transform_homogeneous(
        transform_world2ndc, pos_world);
    const float pos_scrn[3] = {pos_ndc[0], pos_ndc[1], 1.f};
    const auto pos_pix = mat2x3_col_major::mult_vec3(ndc2pix.data(), pos_scrn);
    const cuda::std::array<float,3> sig = mat2_sym::projected_spd_mat3(
        world2pix.data(),
        pnt2splat3[i_pnt].quaternion,
        pnt2splat3[i_pnt].scale);
    const cuda::std::array<float,3> sig_inv = mat2_sym::safe_inverse_preserve_positive_definiteness(sig.data(), 1.0e-5f);
    const cuda::std::array<float,4> _aabb0 = mat2_sym::aabb2(sig_inv.data());
    const cuda::std::array<float,4> _aabb1 = aabb2::scale(_aabb0.data(), 3.f);
    const cuda::std::array<float,4> aabb = aabb2::translate(_aabb1.data(), pos_pix.data());
    // printf("%d %lf %lf %lf\n", i_pnt, sig_inv[0], sig_inv[1], sig_inv[2]);
    // printf("%d %f %f %f %f\n", i_pnt, aabb[0], aabb[1], aabb[2], aabb[3]);
    //
    pnt2splat2[i_pnt].ndc_z = pos_ndc[2];
    pnt2splat2[i_pnt].pos_pix[0] = pos_pix[0];
    pnt2splat2[i_pnt].pos_pix[1] = pos_pix[1];
    pnt2splat2[i_pnt].sig_inv[0] = sig_inv[0];
    pnt2splat2[i_pnt].sig_inv[1] = sig_inv[1];
    pnt2splat2[i_pnt].sig_inv[2] = sig_inv[2];
    pnt2splat2[i_pnt].aabb[0] = aabb[0];
    pnt2splat2[i_pnt].aabb[1] = aabb[1];
    pnt2splat2[i_pnt].aabb[2] = aabb[2];
    pnt2splat2[i_pnt].aabb[3] = aabb[3];
    pnt2splat2[i_pnt].rgb[0] = pnt2splat3[i_pnt].rgb_dc[0];
    pnt2splat2[i_pnt].rgb[1] = pnt2splat3[i_pnt].rgb_dc[1];
    pnt2splat2[i_pnt].rgb[2] = pnt2splat3[i_pnt].rgb_dc[2];
/*

   pnt2splat[i_pnt].rad = rad;
   pnt2splat[i_pnt].rgb[0] = float(pnt2xyzrgb[i_pnt].rgb[0]) / 255.0;
   pnt2splat[i_pnt].rgb[1] = float(pnt2xyzrgb[i_pnt].rgb[1]) / 255.0;
   pnt2splat[i_pnt].rgb[2] = float(pnt2xyzrgb[i_pnt].rgb[2]) / 255.0;
*/
}


}