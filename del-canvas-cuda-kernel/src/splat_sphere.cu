#include "hip/hip_runtime.h"
#include "mat4_col_major.h"
#include "aabb2.h"

extern "C" {

struct Splat3{
    float xyz[3];
    unsigned char rgb[3];
};

struct Splat2 {
    float z;
    float pos_pix[2];
    float rad;
    float rgb[3];
};

__global__
void splat3_to_splat2(
  uint32_t num_pnt,
  Splat2* pnt2splat,
  const Splat3 *pnt2xyzrgb,
  const float *transform_world2ndc,
  const uint32_t img_w,
  const uint32_t img_h,
  float radius)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const auto p0 = pnt2xyzrgb[i_pnt].xyz;
    const auto q0 = mat4_col_major::transform_homogeneous(
        transform_world2ndc, p0);
   float r0[2] = {
     (q0[0] + 1.f) * 0.5f * float(img_w),
     (1.f - q0[1]) * 0.5f * float(img_h) };
   float rad;
   {
       const cuda::std::array<float,9> dqdp = mat4_col_major::jacobian_transform(transform_world2ndc, p0);
       const cuda::std::array<float,9> dpdq = mat3_col_major::try_inverse(dqdp.data()).value();
       const float dx[3] = { dpdq[0], dpdq[1], dpdq[2] };
       const float dy[3] = { dpdq[3], dpdq[4], dpdq[5] };
       float rad_pix_x = (1.f / vec3::norm(dx)) * 0.5f * float(img_w) * radius;
       float rad_pxi_y = (1.f / vec3::norm(dy)) * 0.5f * float(img_h) * radius;
       rad = 0.5f * (rad_pix_x + rad_pxi_y);
   }
   pnt2splat[i_pnt].z = q0[2];
   pnt2splat[i_pnt].pos_pix[0] = r0[0];
   pnt2splat[i_pnt].pos_pix[1] = r0[1];
   pnt2splat[i_pnt].rad = rad;
   pnt2splat[i_pnt].rgb[0] = float(pnt2xyzrgb[i_pnt].rgb[0]) / 255.0;
   pnt2splat[i_pnt].rgb[1] = float(pnt2xyzrgb[i_pnt].rgb[1]) / 255.0;
   pnt2splat[i_pnt].rgb[2] = float(pnt2xyzrgb[i_pnt].rgb[2]) / 255.0;
}


__global__
void count_splat_in_tile(
  uint32_t num_pnt,
  const Splat2* pnt2splat,
  uint32_t* tile2ind,
  uint32_t* pnt2ind,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat2& splat = pnt2splat[i_pnt];
    const cuda::std::array<float,4> aabb = aabb2::from_point(splat.pos_pix, splat.rad);
    //
    float tile_size_f = float(tile_size);
    int ix0 = int(floor(aabb[0] / tile_size_f));
    int iy0 = int(floor(aabb[1] / tile_size_f));
    int ix1 = int(floor(aabb[2] / tile_size_f))+1;
    int iy1 = int(floor(aabb[3] / tile_size_f))+1;
    uint32_t cnt = 0;
    // printf("%d %d %d %d\n", ix0, iy0, ix1, iy1);
    for(int ix = ix0; ix < ix1; ++ix ) {
        if( ix < 0 || ix >= tile_w ){
            continue;
        }
        for(int iy=iy0;iy<iy1;++iy) {
            if( iy < 0 || iy >= tile_h ){
                continue;
            }
            int i_tile = iy * tile_w + ix;
            // printf("%d %d\n", i_pnt, i_tile);
            atomicAdd(&tile2ind[i_tile], 1);
            ++cnt;
        }
    }
    pnt2ind[i_pnt] = cnt;
}

__device__ uint32_t float_to_uint32(float value) {
    uint32_t result;
    memcpy(&result, &value, sizeof(result));
    return result;
}

__device__ uint64_t concatenate32To64(uint32_t a, uint32_t b) {
    return ((uint64_t)b) | (((uint64_t)a) << 32);
}

__global__
void fill_index_info(
  uint32_t num_pnt,
  const Splat2* pnt2splat,
  const uint32_t* pnt2idx,
  uint64_t* idx2tiledepth,
  uint32_t* idx2pnt,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat2& splat = pnt2splat[i_pnt];
    const cuda::std::array<float,4> aabb = aabb2::from_point(splat.pos_pix, splat.rad);
    //
    float tile_size_f = float(tile_size);
    int ix0 = int(floor(aabb[0] / tile_size_f));
    int iy0 = int(floor(aabb[1] / tile_size_f));
    int ix1 = int(floor(aabb[2] / tile_size_f))+1;
    int iy1 = int(floor(aabb[3] / tile_size_f))+1;
    uint32_t cnt = 0;
    // printf("%d %d %d %d\n", ix0, iy0, ix1, iy1);
    for(int ix = ix0; ix < ix1; ++ix ) {
        if( ix < 0 || ix >= tile_w ){
            continue;
        }
        for(int iy=iy0;iy<iy1;++iy) {
            if( iy < 0 || iy >= tile_h ){
                continue;
            }
            uint32_t i_tile = iy * tile_w + ix;
            uint32_t zi = float_to_uint32(splat.z);
            {  // making the negative float value comparable
                zi &= ~(1 << 31); // set zero to 31st bit
                zi = ~zi; // invert bit
            }
            uint64_t tiledepth= concatenate32To64(i_tile, zi);
            idx2tiledepth[pnt2idx[i_pnt] + cnt] = tiledepth;
            idx2pnt[pnt2idx[i_pnt] + cnt] = i_pnt;
            ++cnt;
        }
    }
    // pnt2ind[i_pnt] = cnt;
}

__global__
void rasterize_splat_using_tile(
    uint32_t img_w,
    uint32_t img_h,
    float* d_pix2rgb,
    uint32_t tile_w,
    uint32_t tile_h,
    uint32_t tile_size,
    const uint32_t* d_tile2idx,
    const uint32_t* d_idx2pnt,
    const Splat2* d_pnt2splat)
{
    const uint32_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    if( ix >= img_w ){ return; }
    //
    const uint32_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    if( iy >= img_h ){ return; }
    const uint32_t i_pix = iy * img_w + ix;
    //
    const uint32_t i_tile = (iy / tile_size) * tile_w + (ix / tile_size);
    for(uint32_t idx=d_tile2idx[i_tile]; idx<d_tile2idx[i_tile+1];++idx){
        const uint32_t i_pnt = d_idx2pnt[idx];
        const Splat2& splat = d_pnt2splat[i_pnt];
        const float p0[2] = {
            float(ix) + 0.5f,
            float(iy) + 0.5f};
        const float dx = splat.pos_pix[0] - p0[0];
        const float dy = splat.pos_pix[1] - p0[1];
        const float distance = sqrt(dx * dx + dy * dy);
        if( distance > splat.rad ){ continue; }
        d_pix2rgb[i_pix*3+0] = splat.rgb[0];
        d_pix2rgb[i_pix*3+1] = splat.rgb[1];
        d_pix2rgb[i_pix*3+2] = splat.rgb[2];
    }

}


} // extern "C"