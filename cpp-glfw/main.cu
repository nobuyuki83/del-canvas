#include "hip/hip_runtime.h"
#include <string>
#include <cstdlib>
#include <glad/glad.h>
#define GL_SILENCE_DEPRECATION
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
//
#include "util_opengl.h"
#define M_PI 3.1415

// cudaのエラー検出用マクロ
#define EXIT_IF_FAIL(call)                                                 \
  do {                                                                     \
    (call);                                                                \
    hipError_t err = hipGetLastError();                                  \
    if (err != hipSuccess) {                                              \
      std::cout << "error in file " << __FILE__ << " line at " << __LINE__ \
                << ": " << hipGetErrorString(err) << std::endl;           \
      exit(1);                                                             \
    }                                                                      \
  } while (0)

__global__
void kernel(uchar4 *bitmap, int tick) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  // 連続的になるように...
  float theta = tick / 60.0f * 2.0f * M_PI;
  float theta_x = x / 60.0f * 2.0f * M_PI;
  float theta_y = y / 60.0f * 2.0f * M_PI;
  float r = fabs(sin(theta + theta_x));
  float g = fabs(cos(theta + theta_y));
  float b = fabs(sin(theta + theta_x) * cos(theta + theta_y));

  bitmap[offset].x = (unsigned char)(r * 255);
  bitmap[offset].y = (unsigned char)(g * 255);
  bitmap[offset].z = (unsigned char)(b * 255);
  bitmap[offset].w = 255;
}

// フレームバッファの取得に使用
hipGraphicsResource *dev_resource;

#define WIDTH 1024
#define HEIGHT 1024

int main() {
  if (!glfwInit()) { exit(EXIT_FAILURE); }
  // set OpenGL's version (note: ver. 2.1 is very old, but I chose because it's simple)
  ::glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
  ::glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 1);
  GLFWwindow *window = ::glfwCreateWindow(WIDTH, HEIGHT, "task05", nullptr, nullptr);
  if (!window) { // exit if failed to create window
    ::glfwTerminate();
    exit(EXIT_FAILURE);
  }
  ::glfwMakeContextCurrent(window); // working on this window below
  //
  if (!gladLoadGL()) {     // glad: load all OpenGL function pointers
    printf("Something went wrong in loading OpenGL functions!\n");
    exit(-1);
  }

  int shaderProgram;
  {
    const auto vrt_path = std::string(SOURCE_DIR) + "/shader.vert";
    const auto frg_path = std::string(SOURCE_DIR) + "/shader.frag";
    std::string vrt = acg::load_file_as_string(vrt_path.c_str()); // read source code of vertex shader program
    std::string frg = acg::load_file_as_string(frg_path.c_str()); // read source code of fragment shader program
    shaderProgram = acg::create_shader_program(vrt, frg); // compile the shader on GPU
  }

  glDisable(GL_MULTISAMPLE);
  const GLint iloc = glGetUniformLocation(shaderProgram, "time");  // location of variable in the shader program

  //::glClearColor(1, 1, 1, 1);  // set the color to fill the frame buffer when glClear is called.
  //::glEnable(GL_DEPTH_TEST);

    GLuint pbo;
    // バッファを作成
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER,
                 sizeof(char4) * WIDTH * HEIGHT,
                 NULL,
                 GL_DYNAMIC_DRAW);

    // OpenGLのバッファをCudaと共有する設定
    EXIT_IF_FAIL(hipGraphicsGLRegisterBuffer(
        &dev_resource, pbo, cudaGraphicsMapFlagsNone));

    std::cout << dev_resource << std::endl;

  int tick = 0;  // 今何フレーム目?
  while (!::glfwWindowShouldClose(window)) {
    uchar4 *dev_bitmap;
    size_t size;
    EXIT_IF_FAIL(hipGraphicsMapResources(1, &dev_resource, NULL));
    EXIT_IF_FAIL(hipGraphicsResourceGetMappedPointer(
          (void **)&dev_bitmap, &size, dev_resource));

    // カーネル関数を呼ぶ
    dim3 threads(8, 8);                 // 64スレッド/1グリッド
    dim3 grids(WIDTH / 8, HEIGHT / 8);  // 各ピクセルに1スレッドが割り振られる
    kernel<<<grids, threads>>>(dev_bitmap, tick);

    // カーネル関数の終了を待つ
    EXIT_IF_FAIL(hipDeviceSynchronize());

    // リソースの開放
    EXIT_IF_FAIL(hipGraphicsUnmapResources(1, &dev_resource, NULL));

    ++tick;

/*
    ::glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    const auto time = static_cast<float>(glfwGetTime());
    ::glUniform1f(iloc,time);
    ::glMatrixMode(GL_PROJECTION);
    ::glLoadIdentity(); // identity transformation
    ::glMatrixMode(GL_MODELVIEW);
    ::glLoadIdentity(); // identity transformation
    ::glUseProgram(shaderProgram);  // use the shader program from here
    ::glBegin(GL_QUADS); // draw a rectangle that cover the entire screen
    ::glVertex2d(-1,-1);
    ::glVertex2d(+1,-1);
    ::glVertex2d(+1,+1);
    ::glVertex2d(-1,+1);
    ::glEnd();
*/
    glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    ::glfwSwapBuffers(window);
    ::glfwPollEvents();
  }
  // リソースの開放(glutMainLoop()は返らないので、実際は呼ばれない)
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
  glDeleteBuffers(1, &pbo);
  EXIT_IF_FAIL(cudaGLUnregisterBufferObject(pbo));
  EXIT_IF_FAIL(hipGraphicsUnregisterResource(dev_resource));
  //
  ::glfwDestroyWindow(window);
  ::glfwTerminate();
  exit(EXIT_SUCCESS);
}


